// Copyright (c) 2020-2025 Advanced Micro Devices, Inc. All rights reserved.
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include <thrust/detail/config.h>

#include <thrust/device_vector.h>

// If you create a global `thrust::device_vector` with the default allocator,
// you'll get an error during program termination when the memory of the vector
// is freed, as the CUDA runtime cannot be used during program termination.
//
// To get around this, you can create your own allocator which ignores
// deallocation failures that occur because the CUDA runtime is shut down.

extern "C" hipError_t cudaFreeIgnoreShutdown(void* ptr)
{
  hipError_t const err = hipFree(ptr);
  if (hipSuccess == err || hipErrorDeinitialized == err)
  {
    return hipSuccess;
  }
  return err;
}

using device_ignore_shutdown_memory_resource =
  thrust::system::cuda::detail::cuda_memory_resource<hipMalloc, cudaFreeIgnoreShutdown, thrust::cuda::pointer<void>>;

template <typename T>
using device_ignore_shutdown_allocator =
  thrust::mr::stateless_resource_allocator<T, thrust::device_ptr_memory_resource<device_ignore_shutdown_memory_resource>>;

thrust::device_vector<double, device_ignore_shutdown_allocator<double>> d;

int main()
{
  d.resize(25);
}
