#include "hip/hip_runtime.h"
// Copyright (c) 2020-2025 Advanced Micro Devices, Inc. All rights reserved.
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/random.h>
#include <thrust/transform_reduce.h>

#include <cmath>
#include <iomanip>
#include <iostream>

#include "include/host_device.h"

// we could vary M & N to find the perf sweet spot

__host__ __device__ unsigned int hash(unsigned int a)
{
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

struct estimate_pi
{
  __host__ __device__ float operator()(unsigned int thread_id)
  {
    float sum      = 0;
    unsigned int N = 10000; // samples per thread

    unsigned int seed = hash(thread_id);

    // seed a random number generator
    thrust::default_random_engine rng(seed);

    // create a mapping from random numbers to [0,1)
    thrust::uniform_real_distribution<float> u01(0, 1);

    // take N samples in a quarter circle
    for (unsigned int i = 0; i < N; ++i)
    {
      // draw a sample from the unit square
      float x = u01(rng);
      float y = u01(rng);

      // measure distance from the origin
      float dist = sqrtf(x * x + y * y);

      // add 1.0f if (u0,u1) is inside the quarter circle
      if (dist <= 1.0f)
      {
        sum += 1.0f;
      }
    }

    // multiply by 4 to get the area of the whole circle
    sum *= 4.0f;

    // divide by N
    return sum / N;
  }
};

int main(void)
{
  // use 30K independent seeds
  int M = 30000;

  float estimate = thrust::transform_reduce(
    thrust::counting_iterator<int>(0), thrust::counting_iterator<int>(M), estimate_pi(), 0.0f, thrust::plus<float>());
  estimate /= M;

  std::cout << std::setprecision(3);
  std::cout << "pi is approximately " << estimate << std::endl;

  return 0;
}
