#include "hip/hip_runtime.h"
// Copyright (c) 2020-2025 Advanced Micro Devices, Inc. All rights reserved.
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/gather.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>

#include <iostream>
#include <iterator>

// This example demonstrates how to expand an input sequence by
// replicating each element a variable number of times. For example,
//
//   expand([2,2,2],[A,B,C]) -> [A,A,B,B,C,C]
//   expand([3,0,1],[A,B,C]) -> [A,A,A,C]
//   expand([1,3,2],[A,B,C]) -> [A,B,B,B,C,C]
//
// The element counts are assumed to be non-negative integers

template <typename InputIterator1, typename InputIterator2, typename OutputIterator>
OutputIterator expand(InputIterator1 first1, InputIterator1 last1, InputIterator2 first2, OutputIterator output)
{
  using difference_type = typename thrust::iterator_difference<InputIterator1>::type;

  difference_type input_size  = thrust::distance(first1, last1);
  difference_type output_size = thrust::reduce(first1, last1);

  // scan the counts to obtain output offsets for each input element
  thrust::device_vector<difference_type> output_offsets(input_size, 0);
  thrust::exclusive_scan(first1, last1, output_offsets.begin());

  // scatter the nonzero counts into their corresponding output positions
  thrust::device_vector<difference_type> output_indices(output_size, 0);
  thrust::scatter_if(
    thrust::counting_iterator<difference_type>(0),
    thrust::counting_iterator<difference_type>(input_size),
    output_offsets.begin(),
    first1,
    output_indices.begin());

  // compute max-scan over the output indices, filling in the holes
  thrust::inclusive_scan(
    output_indices.begin(), output_indices.end(), output_indices.begin(), thrust::maximum<difference_type>());

  // gather input values according to index array (output = first2[output_indices])
  thrust::gather(output_indices.begin(), output_indices.end(), first2, output);

  // return output + output_size
  thrust::advance(output, output_size);
  return output;
}

template <typename Vector>
void print(const std::string& s, const Vector& v)
{
  using T = typename Vector::value_type;

  std::cout << s;
  thrust::copy(v.begin(), v.end(), std::ostream_iterator<T>(std::cout, " "));
  std::cout << std::endl;
}

int main(void)
{
  int counts[] = {3, 5, 2, 0, 1, 3, 4, 2, 4};
  int values[] = {1, 2, 3, 4, 5, 6, 7, 8, 9};

  size_t input_size  = sizeof(counts) / sizeof(int);
  size_t output_size = thrust::reduce(counts, counts + input_size);

  // copy inputs to device
  thrust::device_vector<int> d_counts(counts, counts + input_size);
  thrust::device_vector<int> d_values(values, values + input_size);
  thrust::device_vector<int> d_output(output_size);

  // expand values according to counts
  expand(d_counts.begin(), d_counts.end(), d_values.begin(), d_output.begin());

  std::cout << "Expanding values according to counts" << std::endl;
  print(" counts ", d_counts);
  print(" values ", d_values);
  print(" output ", d_output);

  return 0;
}
